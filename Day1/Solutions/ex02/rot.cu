#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <omp.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define MAX_THR 1024

/***
 * coords structure
 ***/
typedef struct {
  float x;
  float y;
} coords;

/***
 * Print usage
 ***/
void
usage(char *argv[])
{
  fprintf(stderr, "usage: %s N\n", argv[0]);
  return;
}

/***
 * Allocate memory; print error if NULL is returned
 ***/
void *
ualloc(size_t size)
{
  void *ptr = malloc(size);
  if(ptr == NULL) {
    fprintf(stderr, "malloc() returned null; quitting...\n");
    exit(-2);
  }
  return ptr;
}

/***
 * Allocate memory on GPU; print error if not successful
 ***/
void *
gpu_alloc(size_t size)
{
  void *ptr;
  hipError_t err = hipMalloc(&ptr, size);
  if(err != hipSuccess) {
    fprintf(stderr, "cudaMalloc() returned %d; quitting...\n", err);
    exit(-2);
  } 
  return ptr;
}

/***
 * Return seconds elapsed since t0, with t0 = 0 the epoch
 ***/
double
stop_watch(double t0)
{
  struct timeval t;
  gettimeofday(&t, NULL);
  return (double)t.tv_sec + (double)t.tv_usec/1e6 - t0;
}

/***
 * Read coords from binary file
 ***/
void
read_coords(coords *r, unsigned long int n, const char *fname)
{
  FILE *fp = fopen(fname, "r");
  for(int i=0; i<n; i++) {
    fread(&r[i].x, sizeof(float), 2, fp);
  }
  fclose(fp);
  return;
}

/***
 * Write coords to binary file
 ***/
void
write_coords(const char *fname, coords *r, unsigned long int n)
{
  FILE *fp = fopen(fname, "w");
  for(int i=0; i<n; i++) {
    fwrite(&r[i].x, sizeof(float), 2, fp);
  }
  fclose(fp);
  return;
}

/***
 * Do r' <- U*r + s on the CPU using OpenMP
 ***/
void
rotate(int n, coords *out, float theta, coords *r, coords *s)
{
  float ct = cos(theta);
  float st = sin(theta);
#pragma omp parallel for
  for(int i=0; i<n; i++) {
    out[i].x = ct*r[i].x - st*r[i].y + s[i].x;
    out[i].y = st*r[i].x + ct*r[i].y + s[i].y;
  }
  return;
}

/***
 * Do r' <- U*r + s on the GPU
 ***/
__global__ void
gpu_rotate(int n, float *out, float theta, float *r, float *s)
{
  float ct = cos(theta);
  float st = sin(theta);

  int ithr = threadIdx.x;
  int nthr = blockDim.x;
  int iblk = blockIdx.x;
  int idx = ithr + iblk*nthr;
  __shared__ float rr[MAX_THR];
  rr[ithr] = r[idx];
  
  float rs = s[idx] + ct*rr[ithr];
  int sw = 1 - 2*(ithr & 1);

  __syncthreads();
  rs = rs - sw*st*rr[ithr+sw];

  out[idx] = rs;
  return;
}


int
main(int argc, char *argv[])
{
  /*
   * If number of arguments are not as expected, print usage and exit
   */
  if(argc != 2) {
    usage(argv);
    return 1;
  }

  unsigned long int n = atol(argv[1]);

  coords *r = (coords *)ualloc(sizeof(coords)*n);
  coords *s = (coords *)ualloc(sizeof(coords)*n);
  coords *v0 = (coords *)ualloc(sizeof(coords)*n);
  coords *v1 = (coords *)ualloc(sizeof(coords)*n);

  /*
   * Read from file
   */
  read_coords(r, n, "points.bin");
  read_coords(s, n, "shifts.bin");
    
  /*
   * The angle
   */
  float theta = (M_PI)*(10.0/180.0);
    
  /*
   * A: Run rotate(), return to v0, report performance
   */
  {
    double t0 = stop_watch(0);
    rotate(n, v0, theta, r, s);
    t0 = stop_watch(t0);

    double n_flop = 8;
    double n_io = 6*sizeof(float);
#pragma omp parallel
    {
#pragma omp single
      {
	int nthr = omp_get_num_threads();
	printf(" CPU: nthr = %4d   t0 = %6.4lf sec   P = %7.3lf Gflop/s   B = %7.3lf GB/s\n",
	       nthr, t0, n_flop*n/1e9/t0, n_io*n/1e9/t0);
      }
    }
  }

  /*
   * B: Run rotate(), return to v1, report performance
   */
  {
    float *d_r = (float *)gpu_alloc(2*n*sizeof(float));
    float *d_s = (float *)gpu_alloc(2*n*sizeof(float));
    float *d_v = (float *)gpu_alloc(2*n*sizeof(float));

    hipMemcpy(d_r, r, sizeof(coords)*n, hipMemcpyHostToDevice);
    hipMemcpy(d_s, s, sizeof(coords)*n, hipMemcpyHostToDevice);
    
    double t0 = stop_watch(0);
    int nthr = 128;
    gpu_rotate<<<2*n/nthr, nthr>>>(n, d_v, theta, d_r, d_s);
    hipDeviceSynchronize();
    t0 = stop_watch(t0);

    hipMemcpy(v1, d_v, sizeof(coords)*n, hipMemcpyDeviceToHost);
    
    double n_flop = 8;
    double n_io = 6*sizeof(float);
    printf(" GPU:               t0 = %6.4lf sec   P = %7.3lf Gflop/s   B = %7.3lf GB/s\n",
	       t0, n_flop*n/1e9/t0, n_io*n/1e9/t0);

    hipFree(d_s);
    hipFree(d_v);
    hipFree(d_r);
  }

  /* Compare v1 and v0 */
  double diff = 0;
  double norm = 0;
  for(int i=0; i<n; i++) {
    float dx = v0[i].x-v1[i].x;
    float dy = v0[i].y-v1[i].y;
    diff += dx*dx + dy*dy;
    norm += v0[i].x*v0[i].x;
    norm += v0[i].y*v0[i].y;
  }
  printf(" Diff = %e\n", diff/norm);

  /*
   * Write to file
   */
  write_coords("points-rot.bin", v0, n);


  
  /*
   * Free arrays
   */
  free(r);
  free(s);
  free(v0);
  free(v1);
  return 0;
}
